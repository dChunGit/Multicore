#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 64

__global__ void buckets_global(int* data, int* result, int total) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < total) {
		if (data[index] >= 0 && data[index] <= 99) atomicAdd(result, 1);
		else if (data[index] >= 100 && data[index] <= 199) atomicAdd(result + 1, 1);
		else if (data[index] >= 200 && data[index] <= 299) atomicAdd(result + 2, 1);
		else if (data[index] >= 300 && data[index] <= 399) atomicAdd(result + 3, 1);
		else if (data[index] >= 400 && data[index] <= 499) atomicAdd(result + 4, 1);
		else if (data[index] >= 500 && data[index] <= 599) atomicAdd(result + 5, 1);
		else if (data[index] >= 600 && data[index] <= 699) atomicAdd(result + 6, 1);
		else if (data[index] >= 700 && data[index] <= 799) atomicAdd(result + 7, 1);
		else if (data[index] >= 800 && data[index] <= 899) atomicAdd(result + 8, 1);
		else if (data[index] >= 900 && data[index] <= 999) atomicAdd(result + 9, 1);
	}
}

__global__ void buckets_local(int*data, int* result, int total) {
    extern __shared__ int local[];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < total) {
		if (data[index] >= 0 && data[index] <= 99) atomicAdd(local, 1);
		else if (data[index] >= 100 && data[index] <= 199) atomicAdd(local + 1, 1);
		else if (data[index] >= 200 && data[index] <= 299) atomicAdd(local + 2, 1);
		else if (data[index] >= 300 && data[index] <= 399) atomicAdd(local + 3, 1);
		else if (data[index] >= 400 && data[index] <= 499) atomicAdd(local + 4, 1);
		else if (data[index] >= 500 && data[index] <= 599) atomicAdd(local + 5, 1);
		else if (data[index] >= 600 && data[index] <= 699) atomicAdd(local + 6, 1);
		else if (data[index] >= 700 && data[index] <= 799) atomicAdd(local + 7, 1);
		else if (data[index] >= 800 && data[index] <= 899) atomicAdd(local + 8, 1);
		else if (data[index] >= 900 && data[index] <= 999) atomicAdd(local + 9, 1);
	}
    __syncthreads();

    int base_index = blockIdx.x * 10;
    for (int i = 0; i < 10; i++) {
    	result[base_index + i] = local[i];
    }
}

__global__ void reduce_buckets_local(int*data, int*result, int num_blocks) {
	int index = blockIdx.x;
	int count = 0;
	for (int i = 0; i < num_blocks; i++) {
		count += data[(10*i) + index];
	}
	result[index] = count;
}


int main(int argc,char **argv)
{

	// initialization
    vector<int> array;
    int i = 0;

    ifstream file( "inp.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    // structures used by all
    int data_size = sizeof(int)*array.size();
    int result_size = sizeof(int)*10;
    int total = array.size();
    int* data = (int*)malloc(data_size);
    int num_blocks = array.size()/THREADS;
    if (array.size()%THREADS != 0) {
    	num_blocks++;
    }
    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
    }

    int* d_data;
	hipMalloc((void **)&d_data, data_size);
    hipMemcpy(d_data, data, data_size, hipMemcpyHostToDevice);

    // 2A
    int* result1 = (int*)malloc(result_size);
    for (int i = 0; i < 10; i++) {
    	result1[i] = 0;
    }
    int* d_result1;
    hipMalloc((void **)&d_result1, result_size);
    hipMemcpy(d_result1, result1, result_size, hipMemcpyHostToDevice);

    buckets_global<<<num_blocks, THREADS>>>(d_data, d_result1, total);

    hipMemcpy(result1, d_result1, result_size, hipMemcpyDeviceToHost);

    // 2B
    int* result_inter = (int*)malloc(result_size*num_blocks);
    for (int i = 0; i < 10*num_blocks; i++) {
    	result_inter[i] = 0;
    }
    int* d_result_inter;
    hipMalloc((void **)&d_result_inter, result_size*num_blocks);
    hipMemcpy(d_result_inter, result_inter, result_size*num_blocks, hipMemcpyHostToDevice);
    
    int* result2 = (int*)malloc(result_size);
    for (int i = 0; i < 10; i++) {
    	result2[i] = 0;
    }
    int* d_result2;
    hipMalloc((void **)&d_result2, result_size);
    hipMemcpy(d_result2, result2, result_size, hipMemcpyHostToDevice);

    buckets_local<<<num_blocks, THREADS, result_size>>>(d_data, d_result_inter, total);

    hipMemcpy(result_inter, d_result_inter, result_size*num_blocks, hipMemcpyDeviceToHost);

    reduce_buckets_local<<<10, 1>>>(d_result_inter, d_result2, num_blocks);

    hipMemcpy(result2, d_result2, result_size, hipMemcpyDeviceToHost);

    //2C


    FILE *fp = fopen("q2a.txt", "w");
    if(fp != NULL) {
        for(int a = 0; a < 9; a++) {
            fprintf(fp, "%d, ", result1[a]);
        }
        fprintf(fp, "%d", result1[9]);
        fclose(fp);
    }

    fp = fopen("q2b.txt", "w");
    if(fp != NULL) {
        for(int a = 0; a < 9; a++) {
            fprintf(fp, "%d, ", result2[a]);
        }
        fprintf(fp, "%d", result2[9]);
        fclose(fp);
    }

    // force the printf()s to flush
    hipDeviceSynchronize();

    return 0;
}
