#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 64

__global__ void buckets_global(int* data, int* result, int total) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < total) {
		if (data[index] >= 0 && data[index] <= 99) atomicAdd(data, 1);
		else if (data[index] >= 100 && data[index] <= 199) atomicAdd(data + 1, 1);
		else if (data[index] >= 200 && data[index] <= 299) atomicAdd(data + 2, 1);
		else if (data[index] >= 300 && data[index] <= 399) atomicAdd(data + 3, 1);
		else if (data[index] >= 400 && data[index] <= 499) atomicAdd(data + 4, 1);
		else if (data[index] >= 500 && data[index] <= 599) atomicAdd(data + 5, 1);
		else if (data[index] >= 600 && data[index] <= 699) atomicAdd(data + 6, 1);
		else if (data[index] >= 700 && data[index] <= 799) atomicAdd(data + 7, 1);
		else if (data[index] >= 800 && data[index] <= 899) atomicAdd(data + 8, 1);
		else if (data[index] >= 900 && data[index] <= 999) atomicAdd(data + 9, 1);
	}
}

__global__ void buckets_local(int*data, int* result, int num_index) {

}


int main(int argc,char **argv)
{

	// initialization
    vector<int> array;
    int i = 0;

    ifstream file( "inp.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    // structures used by all
    int size = sizeof(int)*array.size();
    int total = array.size();
    int* data = (int*)malloc(size);
    int num_blocks = array.size()/THREADS;
    if (array.size()%THREADS != 0) {
    	num_blocks++;
    }
    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
    }

    int* d_data;
	hipMalloc((void **)&d_data, size);
    cudaMemCpy(d_data, data, size, cudaMemCpyHostToDevice);

    // 2A
    int* result1 = (int*)malloc(10*sizeof(int));
    for (int i = 0; i < 10; i++) {
    	result1[i] = 0;
    }
    int* d_result1;
    hipMalloc((void **)&d_result1, 10);
    cudaMemCpy(d_result1, result1, 10, cudaMemCpyHostToDevice);

    buckets_global<<<num_blocks, THREADS>>>(d_data, d_result1, total);

    cudaMemCpy(results1, d_result1, size, hipMemcpyDeviceToHost);

    // 2B
    int* result2 = (int*)malloc(10*sizeof(int)*num_blocks);
    for (int i = 0; i < 10; i++) {
    	result[i] = 0;
    }
    int* d_result2;
    hipMalloc((void **)&d_result2, 10);
    cudaMemCpy(d_result2, result2, 10, cudaMemCpyHostToDevice);

    buckets_local<<<NUM_BLOCKS, 1>>>(d_data, d_result2, total);


    FILE *file = fopen("q2a.txt", "w");
    if(file != NULL) {
        for(int a = 0; a < 9; a++) {
            fprintf(fp, "%d, ", result1[a]);
        }
        fprintf(fp, "%d", result1[9]);
        fclose(file);
    }

    // force the printf()s to flush
    hipDeviceSynchronize();

    return 0;
}
