#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
using namespace std;

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello() {
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc,char **argv) {
    vector<int> array;
    int x, i = 0;

    FILE* inFile = fopen(file1, "r");
    if(inFile != NULL) {
        while(fscanf(inFile, "%lf", &x) != EOF) {
            array[i] = x;
            i++;
        }
    }
    fclose(inFile);
    
    for(int a = 0; a < i - 1; a++) {
        printf("%d, ", array[a]);
    }

    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}
