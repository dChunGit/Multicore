
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
// #include <cuda.h>

using namespace std;

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void setup(int* small) {
    small[blockIdx.x] = 0;
    if(blockIdx.x == 1) {
        small[blockIdx.x] = 1;
    }
    __syncthreads();
}

__global__ void min(int* small, int* data) {
    printf("Thread %d read %d with value %d \n", blockIdx.x, data[blockIdx.x], small[blockIdx.x]);
    __syncthreads();
}

__global__ void finish(int* small, int* data, int* max) {
    if(small[blockIdx.x] == 1) {
        *max = data[blockIdx.x];
    }
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp2.txt" );
    int number;
    char delimiter;
    while((file >> number >> delimiter) && (delimiter == ',')) {
        // use number which has been read 
        array.push_back(number);
        i++;
    }

    int* data = new int[array.size()];
    int* d_data;
    int* d_small;
    int max;
    int* d_max;
    copy(array.begin(), array.end(), data);
    int size = sizeof(int)*array.size();

    hipMalloc((void **) &d_data, size);
    hipMalloc((void **) &d_max, sizeof(int));
    hipMalloc((void **) &d_small, size);

    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    // launch the kernel
    setup<<<array.size(), BLOCK_WIDTH>>>(d_small);
    min<<<array.size(), BLOCK_WIDTH>>>(d_small, d_data);
    finish<<<array.size(), BLOCK_WIDTH>>>(d_small, d_data, d_max);

    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_max);

    // force the printf()s to flush
    hipDeviceSynchronize();

    return 0;
}
