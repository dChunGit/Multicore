
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
// #include <cuda.h>

using namespace std;

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void min(int* iB, int* max) {
    printf("Thread %d read %d\n", blockIdx.x, iB[blockIdx.x]);
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp2.txt" );
    int number;
    char delimiter;
    while((file >> number >> delimiter) && (delimiter == ',')) {
        // use number which has been read 
        array.push_back(number);
        i++;
    }

    int* iB = new int[array.size()];
    int* d_iB;
    int max;
    int * d_max;
    copy(array.begin(), array.end(), iB);
    for(int a = 0; a < array.size(); a++) {
        printf("%d ", iB[a]);
    }

    int size = sizeof(int)*array.size();
    printf("%d", size);

    hipMalloc((void **) &d_iB, size);
    hipMalloc((void **) &d_max, sizeof(int));

    hipMemcpy(d_iB, iB, size, hipMemcpyHostToDevice);

    // launch the kernel
    min<<<array.size(), BLOCK_WIDTH>>>(d_iB, d_max);

    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_iB);
    hipFree(d_max);

    // force the printf()s to flush
    hipDeviceSynchronize();

    return 0;
}
