
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
using namespace std;

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void min() {
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}


int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp2.txt" );
    int number;
    char delimiter;
    while((file >> number >> delimiter) && (delimiter == ',')) {
        // use number which has been read 
        printf("%d ", number);
        array.push_back(number);
        i++;
    }

    // launch the kernel
    min<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    return 0;
}
