#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 256

__global__ void setup(int* small, int n) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < n) {
        small[thid] = 1;
    }
}

__global__ void min(int* small, int* data, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for(int j = 0; j < n; j++) {
        if(i < n) {
            if(data[j] < data[i] && i != j) {
                small[i] = 0;
            }
        }
    }
}

__global__ void finish(int* small, int* data, int* max, int n) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < n) {
        if(small[thid] == 1) {
            *max = data[thid];
        }
    }
}

__global__ void last_digits(int* mod, int* data, int n) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < n) {
        mod[thid] = data[thid]%10;
    }
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    int* data = new int[array.size()];
    int* mod = new int[array.size()];
    int* d_data;
    int* d_small;
    int max;
    int* d_max;
    int* d_mod;
    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
    }

    int size = sizeof(int)*array.size();

    hipMalloc((void **) &d_data, size);
    hipMalloc((void **) &d_max, sizeof(int));
    hipMalloc((void **) &d_small, size);
    hipMalloc((void **) &d_mod, size);

    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    int sizing = array.size()/THREADS;
    if(array.size()%THREADS > 0) {
        sizing++;
    }
    // launch the kernel
    setup<<<sizing, THREADS>>>(d_small, (int) array.size());
    min<<<sizing, THREADS>>>(d_small, d_data, (int) array.size());
    finish<<<sizing, THREADS>>>(d_small, d_data, d_max, (int) array.size());
    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    last_digits<<<sizing, THREADS>>>(d_mod, d_data, array.size());
    hipMemcpy(mod, d_mod, size, hipMemcpyDeviceToHost);

    FILE *fp = fopen("q1b.txt", "w");
    if(fp != NULL) {
        for(int a = 0; a < array.size() - 1; a++) {
            fprintf(fp, "%d, ", mod[a]);
        }
        fprintf(fp, "%d", mod[array.size() - 1]);
        fclose(fp);
    }

    fp = fopen("q1a.txt", "w");
    if(fp != NULL) {
        fprintf(fp, "%d", max);
        fclose(fp);
    }


    printf("\n");
    printf("%d ", max);
    printf("\n");

    hipFree(d_data);
    hipFree(d_max);
    hipFree(d_mod);

    // force the printf()s to flush
    hipDeviceSynchronize();


    return 0;
}
