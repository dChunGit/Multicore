#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 20

__global__ void setup(int* data, int* odd, int* count, int n) {
	int thid = blockIdx.x*THREADS + threadIdx.x;
	if(thid >= n) {
		return;
	}
	
	if(data[thid]%2 == 1) {
	    odd[thid] = 1;
	    atomicAdd(count, 1);
	} else odd[thid] = 0;
}

__global__ void parallelPrefix(int* odd, int* ppref, int n) {
	int thid = blockIdx.x*THREADS + threadIdx.x;
	if(thid >= n) {
		return;
	}
	int val = 0;
	ppref[thid] = odd[thid];
	printf("%d ", ppref[thid]);
	__syncthreads();

	for(int i = 1; i < n; i *= 2) {
		if(thid >= i) {
			val = ppref[thid - i];
		}
		__syncthreads();

		if(thid >= i) {
			ppref[thid] += val;
		}
		__syncthreads();
	}
}

__global__ void finish(int* odd, int* ppref, int* results, int* data, int n) {
	int thid = blockIdx.x*THREADS + threadIdx.x;
	if(thid >= n) {
		return;
	}
	
	if(odd[thid] == 1) {
		results[ppref[thid] - 1] = data[thid];
	}
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp2.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    int* data = new int[array.size()];
    int* ppref = new int[array.size()];
    int* odd = new int[array.size()];
    int count;
    int* d_data;
    int* d_odd;
    int* d_ppref;
    int* d_count;

    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
        printf("%d ", data[a]);
    }
    printf("\n");

    int size = sizeof(int)*array.size();

    hipMalloc((void **) &d_data, size);
    hipMalloc((void **) &d_odd, size);
    hipMalloc((void **) &d_ppref, size);
    hipMalloc((void **) &d_count, sizeof(int));

    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);  

    // launch the kernel
    int blocks = array.size()/THREADS;
    if(array.size()%THREADS > 0) {
    	blocks += 1;
    }

    setup<<<blocks, THREADS>>>(d_data, d_odd, d_count, array.size());
    //get number of odds
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    //do parallel prefix on odd array to find distance from the start
    parallelPrefix<<<blocks, THREADS>>>(d_odd, d_ppref, array.size());
    // force the printf()s to flush
    hipDeviceSynchronize();

    hipMemcpy(ppref, d_ppref, size, hipMemcpyDeviceToHost); 
    printf("\n");
    for(int a = 0; a < array.size(); a++) {
    	printf("(%d:%d), ", ppref[a], data[a]);
    }

    //create array, if odd from small, copy into location of array
    // int* results = new int[count];
    // int* d_results;
    // cudaMalloc((void **) &d_results, sizeof(int)*count);
    // finish<<<blocks, THREADS>>>(d_odd, d_ppref, d_results, d_data, array.size());
    // cudaMemcpy(results, d_results, sizeof(int)*count, cudaMemcpyDeviceToHost);
    // printf("\n");
    // for(int a = 0; a < count; a++) {
    //     printf("%d ", results[a]);
    // }

    // cudaFree(d_data);
    // cudaFree(d_odd);
    hipFree(d_ppref);
    hipFree(d_count);
    // cudaFree(d_results);

    // force the printf()s to flush
    hipDeviceSynchronize();


    return 0;
}
