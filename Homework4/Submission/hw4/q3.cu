#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 64

__global__ void setup(int* data, int* odd, int* count, int n) {
	int thid = blockIdx.x*blockDim.x + threadIdx.x;
	if(thid >= n) {
		return;
	}
	
	if(data[thid]%2 == 1) {
	    odd[thid] = 1;
	    atomicAdd(count, 1);
	} else odd[thid] = 0;
}

__global__ void parallelPrefix(int* odd, int* ppref, int* offset, int n) {
	extern __shared__ int prefs[];
	int thid = blockIdx.x*blockDim.x + threadIdx.x;
	if(thid >= n) {
		return;
	}
	prefs[threadIdx.x] = odd[thid];

	__syncthreads();

	int val = 0;
	for(int i = 1; i < THREADS; i *= 2) {
		if(threadIdx.x >= i) {
			val = prefs[threadIdx.x - i];
		}
		__syncthreads();

		if(threadIdx.x >= i) {
			prefs[threadIdx.x] += val;
		}
		__syncthreads();
	}
	ppref[thid] = prefs[threadIdx.x];
	if(threadIdx.x == THREADS - 1) {
		offset[blockIdx.x] = ppref[thid];
	}
}

__global__ void sum_reduce(int* offset, int n) {
	extern __shared__ int prefs[];
	int thid = threadIdx.x;

	printf("\n");
	prefs[thid] = offset[thid];
	__syncthreads();

	int val = 0;
	for(int i = 1; i < n; i *= 2) {
		if(thid >= i) {
			val = prefs[thid - i];
		}
		// printf("%d \n", val);
		__syncthreads();

		if(thid >= i) {
			prefs[thid] += val;
		}
		__syncthreads();
	}
	offset[thid] = prefs[thid];
}


__global__ void concat(int* offset, int*ppref) {
	int thid = blockIdx.x*blockDim.x + threadIdx.x;
	if(blockIdx.x > 0) {
		ppref[thid] += offset[blockIdx.x - 1];
	}
}

__global__ void finish(int* odd, int* ppref, int* results, int* data, int n) {
	int thid = blockIdx.x*blockDim.x + threadIdx.x;
	if(thid >= n) {
		return;
	}
	
	if(odd[thid] == 1) {
		results[ppref[thid] - 1] = data[thid];
	}
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    int* data = new int[array.size()];
    int* ppref = new int[array.size()];
    int* odd = new int[array.size()];
    int count;
    int* d_data;
    int* d_odd;
    int* d_ppref;
    int* d_count;
    int* d_offset;

    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
    }
    printf("\n");

    int size = sizeof(int)*array.size();

    hipMalloc((void **) &d_data, size);
    hipMalloc((void **) &d_odd, size);
    hipMalloc((void **) &d_ppref, size);
    hipMalloc((void **) &d_count, sizeof(int));

    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);  

    // launch the kernel
    int blocks = array.size()/THREADS;
    if(array.size()%THREADS > 0) {
    	blocks += 1;
    }
    hipMalloc((void **) &d_offset, sizeof(int)*blocks);
    int* offset = new int[blocks];

    setup<<<blocks, THREADS>>>(d_data, d_odd, d_count, array.size());
    //get number of odds
    hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    //do parallel prefix on odd array to find distance from the start
    parallelPrefix<<<blocks, THREADS, sizeof(int)*THREADS>>>(d_odd, d_ppref, d_offset, array.size());
    hipMemcpy(offset, d_offset, sizeof(int)*blocks, hipMemcpyDeviceToHost); 

    sum_reduce<<<1, blocks, sizeof(int)*blocks>>>(d_offset, blocks);
    concat<<<blocks, THREADS>>>(d_offset, d_ppref);

    hipMemcpy(ppref, d_ppref, size, hipMemcpyDeviceToHost); 

    //create array, if odd from small, copy into location of array
    int* results = new int[count];
    int* d_results;
    hipMalloc((void **) &d_results, sizeof(int)*count);
    finish<<<blocks, THREADS>>>(d_odd, d_ppref, d_results, d_data, array.size());
    hipMemcpy(results, d_results, sizeof(int)*count, hipMemcpyDeviceToHost);
    // printf("\n");
    // for(int a = 0; a < count; a++) {
    //     printf("%d ", results[a]);
    // }

    FILE *fp = fopen("q3.txt", "w");
    if(fp != NULL) {
        for(int a = 0; a < count; a++) {
            fprintf(fp, "%d", results[a]);
            if(a + 1 < count) {
                fprintf(fp, ", ");
            }
        }
        fclose(fp);
    }

    hipFree(d_data);
    hipFree(d_odd);
    hipFree(d_ppref);
    hipFree(d_count);
    hipFree(d_results);

    // force the printf()s to flush
    hipDeviceSynchronize();


    return 0;
}
