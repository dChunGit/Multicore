#include <stdio.h>
#include <vector>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <hip/hip_runtime.h>

using namespace std;

#define THREADS 64

__global__ void last_digits(int* mod, int* data, int n) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < n) {
        mod[thid] = data[thid]%10;
    }
}

__global__ void min_reduction(int * data, int* results, int n) {
    extern __shared__ int temp[];
    int thid = blockIdx.x*blockDim.x + threadIdx.x;
    int lid = threadIdx.x;

    if(thid < n) {
        temp[lid] = data[thid];
    } else temp[lid] = 1000;
    __syncthreads();

    for(int offset = blockDim.x>>1; offset > 0; offset >>= 1) {
        __syncthreads();
        if(lid < offset) {
            if(temp[lid + offset] < temp[lid]) {
                temp[lid] = temp[lid + offset];
            }
        }
    }

    if(lid == 0) {
        results[blockIdx.x] = temp[0];
    }
}

int main(int argc,char **argv) {
    vector<int> array;
    int i = 0;

    ifstream file( "inp.txt" );
    int number;
    while(file>>number) {
        array.push_back(number); 
        i++;
        if (file.peek() == ',')
            file.ignore();
    }

    int* data = new int[array.size()];
    int* mod = new int[array.size()];
    int* d_data;
    int* d_mod;
    for(int a = 0; a < array.size(); a++) {
        data[a] = array[a];
    }

    int size = sizeof(int)*array.size();

    hipMalloc((void **) &d_data, size);
    hipMalloc((void **) &d_mod, size);

    hipMemcpy(d_data, data, size, hipMemcpyHostToDevice);

    int sizing = array.size()/THREADS;
    if(array.size()%THREADS > 0) {
        sizing++;
    }

    int* inter = new int[array.size()];
    int* d_inter;
    int blockSize = sizeof(int)*THREADS;
    hipMalloc((void **) &d_inter, size);

    // first reduction
    min_reduction<<<sizing, THREADS, blockSize>>>(d_data, d_inter, array.size());
    hipMemcpy(inter, d_inter, size, hipMemcpyDeviceToHost);

    // second reduction
    int* results = new int[array.size()];
    int* d_results;
    hipMalloc((void **) &d_results, size);
    
    min_reduction<<<sizing, THREADS, blockSize>>>(d_inter, d_results, sizing);
    hipMemcpy(results, d_results, blockSize, hipMemcpyDeviceToHost);

    // last digits of array
    last_digits<<<sizing, THREADS>>>(d_mod, d_data, array.size());
    hipMemcpy(mod, d_mod, size, hipMemcpyDeviceToHost);

    FILE *fp = fopen("q1b.txt", "w");
    if(fp != NULL) {
        for(int a = 0; a < array.size(); a++) {
            fprintf(fp, "%d", mod[a]);
            if(a + 1 < array.size()) {
                fprintf(fp, ", ");
            }
        }
        fclose(fp);
    }

    fp = fopen("q1a.txt", "w");
    if(fp != NULL && array.size() > 0) {
        fprintf(fp, "%d", results[0]);
        fclose(fp);
    }

    hipFree(d_data);
    hipFree(d_inter);
    hipFree(d_results);
    hipFree(d_mod);

    // force the printf()s to flush
    hipDeviceSynchronize();


    return 0;
}
